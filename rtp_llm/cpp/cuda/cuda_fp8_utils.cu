#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_fp8_utils.h"
#include "rtp_llm/cpp/cuda/reduce_kernel_utils.cuh"
#include "rtp_llm/cpp/cuda/cuda_type_utils.cuh"
#include <algorithm>
#include <cstdio>
#include <hip/hip_fp16.h>
#include <limits>
#include <type_traits>

using namespace rtp_llm;
namespace tensorrt_llm
{
namespace common
{
#ifdef ENABLE_FP8

constexpr int CTA_SIZE = 256;

template <bool QUANTIZE>
__inline__ __device__ float scale(float a, float b)
{
    return QUANTIZE ? a / b : a * b;
}

template <QuantizeMode QUANTIZE_MODE, bool QUANTIZE, typename T_OUT, typename T_S, typename T_IN>
__global__ void scaleMatrix(T_OUT* output, T_S const* input_scale, T_IN const* input, int64_t numel, int64_t lda)
{
    for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < numel; i += blockDim.x * gridDim.x)
    {

        if (QUANTIZE_MODE == QuantizeMode::PER_CHANNEL)
        {
            output[i] = T_OUT(scale<QUANTIZE>(static_cast<float>(input[i]), static_cast<float>(input_scale[i % lda])));
        }
        else if (QUANTIZE_MODE == QuantizeMode::PER_TOKEN)
        {
            output[i] = T_OUT(scale<QUANTIZE>(static_cast<float>(input[i]), static_cast<float>(input_scale[i / lda])));
        }
        else if (QUANTIZE_MODE == QuantizeMode::PER_TENSOR)
        {
            output[i] = T_OUT(scale<QUANTIZE>(static_cast<float>(input[i]), static_cast<float>(input_scale[0])));
	}     
    }
}

template <typename T_OUT, typename T_S, typename T_IN>
void invokeQuantizeMatrix(T_OUT* output, T_S const* input_scale, T_IN const* input, int64_t numel, int64_t lda,
    QuantizeMode quantize_mode, hipStream_t stream)
{
    dim3 grid(1024);
    dim3 block(CTA_SIZE);
    if (quantize_mode == QuantizeMode::PER_CHANNEL)
    {
        scaleMatrix<QuantizeMode::PER_CHANNEL, true>
            <<<grid, block, 0, stream>>>(output, input_scale, input, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TOKEN)
    {
        scaleMatrix<QuantizeMode::PER_TOKEN, true><<<grid, block, 0, stream>>>(output, input_scale, input, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TENSOR)
    {
        scaleMatrix<QuantizeMode::PER_TENSOR, true><<<grid, block, 0, stream>>>(output, input_scale, input, numel, lda);
    }
    sync_check_cuda_error();
}

template <typename T_OUT, typename T_S, typename T_IN>
void invokeDequantizeMatrix(T_OUT* output, T_S const* input_scale, T_IN const* input, int64_t numel, int64_t lda,
    QuantizeMode quantize_mode, hipStream_t stream)
{
    dim3 grid(1024);
    dim3 block(CTA_SIZE);
    if (quantize_mode == QuantizeMode::PER_CHANNEL)
    {
        scaleMatrix<QuantizeMode::PER_CHANNEL, false>
            <<<grid, block, 0, stream>>>(output, input_scale, input, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TOKEN)
    {
        scaleMatrix<QuantizeMode::PER_TOKEN, false><<<grid, block, 0, stream>>>(output, input_scale, input, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TENSOR)
    {
        scaleMatrix<QuantizeMode::PER_TENSOR, false>
            <<<grid, block, 0, stream>>>(output, input_scale, input, numel, lda);
    }
    sync_check_cuda_error();
}

template <typename T_FAKE, typename T_OUT, typename T_IN>
__global__ void fakeQuantize(T_OUT* dst, const T_IN* src, const int64_t numel)
{
    for (int64_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < numel; tid += blockDim.x * gridDim.x)
    {
        T_FAKE tmp = (T_FAKE) (static_cast<float>(src[tid]));
        dst[tid] = (T_OUT) (static_cast<float>(tmp));
    }
}

template <typename T_FAKE, typename T_OUT, typename T_IN>
void invokeFakeQuantize(T_OUT* dst, const T_IN* src, const int64_t numel, hipStream_t stream)
{
    fakeQuantize<T_FAKE><<<1024, CTA_SIZE, 0, stream>>>(dst, src, numel);
    sync_check_cuda_error();
}

template void invokeFakeQuantize<__hip_fp8_e4m3_fnuz, float, float>(
    float* dst, float const* src, const int64_t numel, hipStream_t stream);
template void invokeFakeQuantize<float, float, __hip_fp8_e4m3_fnuz>(
    float* dst, __hip_fp8_e4m3_fnuz const* src, const int64_t numel, hipStream_t stream);
template void invokeFakeQuantize<__hip_fp8_e4m3_fnuz, half, half>(
    half* dst, half const* src, const int64_t numel, hipStream_t stream);
template void invokeFakeQuantize<__hip_fp8_e4m3_fnuz, __hip_bfloat16, __hip_bfloat16>(
    __hip_bfloat16* dst, __hip_bfloat16 const* src, const int64_t numel, hipStream_t stream);

template void invokeFakeQuantize<float, half, float>(
    half* dst, float const* src, const int64_t numel, hipStream_t stream);

__device__ float atomicMaxExtd(float* address, float val)
{
    assert(val >= 0);
    unsigned int* address_as_u = reinterpret_cast<unsigned int*>(address);
    unsigned int old = atomicMax(address_as_u, __float_as_uint(val));
    return __uint_as_float(old);
}

template <typename T>
inline __device__ T atomicMaxExtdV2(T* address, T val)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    static_assert(std::is_same_v<T, half> | std::is_same_v<T, __hip_bfloat16>, "T needs to be either half or bfloat16");
    // The address in 64 bits.
    uint64_t address_u64 = reinterpret_cast<uint64_t const&>(address);

    // Pack the input value into 32 bits.
    union
    {
        T v[2];
        uint16_t u[2];
    } old, tmp = {};

    int const loc = (address_u64 & 0x2) >> 1;
    tmp.v[loc] = val;

    // 4B aligned pointer.
    auto aligned_address = reinterpret_cast<T*>(address_u64 & ~0x3ull);

    if constexpr (std::is_same_v<T, half>)
    {
        asm volatile("atom.global.v2.f16.max.noftz {%0, %1}, [%2], {%3, %4};"
                     : "=h"(old.u[0]), "=h"(old.u[1])
                     : "l"(aligned_address), "h"(tmp.u[0]), "h"(tmp.u[1]));
    }
    if constexpr (std::is_same_v<T, __hip_bfloat16>)
    {
        asm volatile("atom.global.v2.bf16.max.noftz {%0, %1}, [%2], {%3, %4};"
                     : "=h"(old.u[0]), "=h"(old.u[1])
                     : "l"(aligned_address), "h"(tmp.u[0]), "h"(tmp.u[1]));
    }

    // Return the correct half.
    return old.v[loc];
#endif
}

__device__ half atomicMaxExtd(half* address, half val)
{
    unsigned short int* address_as_u = reinterpret_cast<unsigned short int*>(address);
    unsigned short int old = *address_as_u, assumed;

    while (val > __ushort_as_half(old))
    {
        assumed = old;
        old = atomicCAS(address_as_u, assumed, __half_as_ushort(val));
    }

    return __ushort_as_half(old);
}

__device__ __hip_bfloat16 atomicMaxExtd(__hip_bfloat16* address, __hip_bfloat16 val)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
    unsigned short int* address_as_u = reinterpret_cast<unsigned short int*>(address);
    unsigned short int old = *address_as_u, assumed;

    while (val > __ushort_as_bfloat16(old))
    {
        assumed = old;
        old = atomicCAS(address_as_u, assumed, __bfloat16_as_ushort(val));
    }

    return __ushort_as_bfloat16(old);
#else
    assert(0);
    asm volatile("brkpt;\n" ::);
    return __hip_bfloat16(0);
#endif
}

template <QuantizeMode QUANTIZE_MODE, typename T_S, typename T_W>
__global__ void computeFP8QuantizeScale(T_S* quant_ptr, const T_W* weights, const int64_t size, const int64_t n)
{
    constexpr float min_scaling_factor = 1.0f / (FP8_E4M3_MAX * 512.f);
    if (QUANTIZE_MODE == QuantizeMode::PER_CHANNEL)
    {
        for (int64_t col = threadIdx.x; col < n; col += blockDim.x)
        {
            float max = 0.f;
            for (int64_t i = col + n * blockIdx.x; i < size; i += gridDim.x * n)
            {
                auto val = fabs(static_cast<float>(weights[i]));
                max = max > val ? max : val;
            }
            auto const scale = (T_S) std::max(max / FP8_E4M3_MAX, min_scaling_factor);
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
            if constexpr (std::is_same_v<T_S, float>)
            {
                atomicMaxExtd(quant_ptr + col, scale);
            }
            else
            {
                auto const address_u64 = reinterpret_cast<uint64_t>(quant_ptr + col);
                if ((col == 0 && address_u64 % 4 != 0) || (col == n - 1 && address_u64 % 4 == 0))
                    atomicMaxExtd(quant_ptr + col, scale);
                else
                    atomicMaxExtdV2(quant_ptr + col, scale);
            }
#else // Vector atomics require __CUDA_ARCH__ >= 900
            atomicMaxExtd(quant_ptr + col, scale);
#endif
        }
    }
    else if (QUANTIZE_MODE == QuantizeMode::PER_TOKEN)
    {
        auto const nrows = size / n;
        for (int64_t row = blockIdx.x; row < nrows; row += gridDim.x)
        {
            float max = 0.f;
            for (int64_t i = threadIdx.x; i < n; i += blockDim.x)
            {
                auto val = fabs(static_cast<float>(weights[row * n + i]));
                max = max > val ? max : val;
            }
            max = blockReduceMax<float>(max);
            if (threadIdx.x == 0)
            {
                auto const scale = (T_S) std::max(max / FP8_E4M3_MAX, min_scaling_factor);
                quant_ptr[row] = scale;
            }
        }
    }
    else if (QUANTIZE_MODE == QuantizeMode::PER_TENSOR)
    {
        float max = 0.f;
        for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += gridDim.x * blockDim.x)
        {
            auto val = fabs(static_cast<float>(weights[i]));
            max = max > val ? max : val;
        }
        max = blockReduceMax<float>(max);
        if (threadIdx.x == 0)
        {
            auto const scale = (T_S) std::max(max / FP8_E4M3_MAX, min_scaling_factor);
            atomicMaxExtd(quant_ptr, scale);
        }
    }
}

template <typename T_S, typename T_W>
void invokeComputeFP8QuantizeScale(T_S* quant_ptr, const T_W* weights, const int64_t numel, const int64_t lda,
    QuantizeMode quantize_mode, hipStream_t stream)
{
    if (quantize_mode == QuantizeMode::PER_TOKEN)
    {
        dim3 block(CTA_SIZE);
        dim3 grid(numel / lda);
        computeFP8QuantizeScale<QuantizeMode::PER_TOKEN><<<grid, block, 0, stream>>>(quant_ptr, weights, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_CHANNEL)
    {
        dim3 block(CTA_SIZE);
        dim3 grid((lda + CTA_SIZE - 1) / CTA_SIZE);
        hipMemsetAsync(quant_ptr, 0, lda * sizeof(T_S), stream);
        sync_check_cuda_error();
        computeFP8QuantizeScale<QuantizeMode::PER_CHANNEL><<<grid, block, 0, stream>>>(quant_ptr, weights, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TENSOR)
    {
        dim3 block(1024);
        dim3 grid(1024);
        hipMemsetAsync(quant_ptr, 0, sizeof(T_S), stream);
        sync_check_cuda_error();
        computeFP8QuantizeScale<QuantizeMode::PER_TENSOR><<<grid, block, 0, stream>>>(quant_ptr, weights, numel, lda);
    }
    sync_check_cuda_error();
}

#define DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(type_scale, type_in)                                                  \
    template void invokeComputeFP8QuantizeScale<type_scale, type_in>(type_scale * input_scale, type_in const* weights, \
        int64_t numel, int64_t lda, QuantizeMode quantize_mode, hipStream_t stream);

DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(half, half);
DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(float, half);
DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(float, float);
#ifdef ENABLE_BF16
DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(__hip_bfloat16, __hip_bfloat16);
DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(float, __hip_bfloat16);
#endif

template <typename T_OUT, typename T_S, typename T_IN>
__global__ void dynamicQuantizeMatrixPerToken(
    T_OUT* output, T_S* quant_ptr, T_IN const* input, int64_t numel, int64_t lda)
{
    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T_IN* shmem = reinterpret_cast<T_IN*>(_shmem);
    constexpr float min_scaling_factor = 1.0f / (FP8_E4M3_MAX * 512.f);
    auto const nrows = numel / lda;
    for (int64_t row = blockIdx.x; row < nrows; row += gridDim.x)
    {
        float max = 0.f;
        for (int64_t i = threadIdx.x; i < lda; i += blockDim.x)
        {
            auto const in = input[row * lda + i];
            shmem[i] = in;
            auto val = fabs(static_cast<float>(in));
            max = max > val ? max : val;
        }
        max = blockAllReduceMax<float>(max); // __syncthreads() called so we can read shmem
        auto const s = (T_S) std::max(max / FP8_E4M3_MAX, min_scaling_factor);
        for (int64_t i = threadIdx.x; i < lda; i += blockDim.x)
        {
            // true means we are quantizing
            output[row * lda + i] = (T_OUT) scale<true>(static_cast<float>(shmem[i]), static_cast<float>(s));
        }
        if (threadIdx.x == 0)
        {
            quant_ptr[row] = s;
        }
    }
}

template <typename T_OUT, typename T_S, typename T_IN>
void invokeComputeScalesAndQuantizeMatrix(T_OUT* output, T_S* quant_ptr, const T_IN* input, const int64_t numel,
    const int64_t lda, QuantizeMode quantize_mode, hipStream_t stream)
{
    if (quantize_mode == QuantizeMode::PER_TOKEN)
    {
        dim3 grid(numel / lda);
        bool use_shmem = true;
        auto const shmem_size = lda * sizeof(T_IN);
        if (shmem_size >= (48 << 10))
        {
            hipError_t ret = hipFuncSetAttribute(reinterpret_cast<const void*>(dynamicQuantizeMatrixPerToken<T_OUT), T_S, T_IN>,
                hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size);
            use_shmem = ret == hipSuccess;
        }
        if (use_shmem)
        {
            // ensure the threadblock is as large as possible to increase occupancy
            dim3 block(std::min((lda + 31) / 32 * 32, static_cast<int64_t>(1024)));
            dynamicQuantizeMatrixPerToken<<<grid, block, shmem_size, stream>>>(output, quant_ptr, input, numel, lda);
        }
        else
        {
            dim3 block(CTA_SIZE);
            computeFP8QuantizeScale<QuantizeMode::PER_TOKEN><<<grid, block, 0, stream>>>(quant_ptr, input, numel, lda);
            sync_check_cuda_error();
            invokeQuantizeMatrix(output, quant_ptr, input, numel, lda, quantize_mode, stream);
        }
    }
    else if (quantize_mode == QuantizeMode::PER_CHANNEL)
    {
        dim3 block(CTA_SIZE);
        dim3 grid((lda + CTA_SIZE - 1) / CTA_SIZE);
        hipMemsetAsync(quant_ptr, 0, lda * sizeof(T_S), stream);
        sync_check_cuda_error();
        computeFP8QuantizeScale<QuantizeMode::PER_CHANNEL><<<grid, block, 0, stream>>>(quant_ptr, input, numel, lda);
        sync_check_cuda_error();
        invokeQuantizeMatrix(output, quant_ptr, input, numel, lda, quantize_mode, stream);
    }
    else if (quantize_mode == QuantizeMode::PER_TENSOR)
    {
        dim3 block(1024);
        dim3 grid(1024);
        hipMemsetAsync(quant_ptr, 0, sizeof(T_S), stream);
        sync_check_cuda_error();
        computeFP8QuantizeScale<QuantizeMode::PER_TENSOR><<<grid, block, 0, stream>>>(quant_ptr, input, numel, lda);
        sync_check_cuda_error();
        invokeQuantizeMatrix(output, quant_ptr, input, numel, lda, quantize_mode, stream);
    }
    sync_check_cuda_error();
}

#define DEFINE_INVOKE_QUANTIZE_MATRIX(type_out, type_scale, type_in)                                                   \
    template void invokeQuantizeMatrix<type_out, type_scale, type_in>(type_out * output,                               \
        type_scale const* input_scale, type_in const* input, int64_t numel, int64_t lda, QuantizeMode quantize_mode,   \
        hipStream_t stream);                                                                                          \
    template void invokeDequantizeMatrix<type_out, type_scale, type_in>(type_out * output,                             \
        type_scale const* input_scale, type_in const* input, int64_t numel, int64_t lda, QuantizeMode quantize_mode,   \
        hipStream_t stream);                                                                                          \
    template void invokeComputeScalesAndQuantizeMatrix<type_out, type_scale, type_in>(type_out * output,               \
        type_scale * input_scale, type_in const* input, int64_t numel, int64_t lda, QuantizeMode quantize_mode,        \
        hipStream_t stream);

#ifdef ENABLE_FP8
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, float, float);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, float, half);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, half, half);
DEFINE_INVOKE_QUANTIZE_MATRIX(half, half, __hip_fp8_e4m3_fnuz);
DEFINE_INVOKE_QUANTIZE_MATRIX(float, float, __hip_fp8_e4m3_fnuz);
DEFINE_INVOKE_QUANTIZE_MATRIX(half, float, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, __hip_bfloat16, __hip_bfloat16);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, float, __hip_bfloat16);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_bfloat16, __hip_bfloat16, __hip_fp8_e4m3_fnuz);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_bfloat16, float, __hip_fp8_e4m3_fnuz);
#endif
#endif

#endif // ENABLE_FP8

#ifdef ENABLE_FP8
inline __device__ __hip_bfloat16 max_abs_op(bf16_4_t v) {
    return cuda_max(cuda_max<__hip_bfloat16>(cuda_abs(v.x)), cuda_max<__hip_bfloat16>(cuda_abs(v.y)));
}

inline __device__ __hip_bfloat16 max_abs_op(bf16_8_t v) {
    return cuda_max<__hip_bfloat16>(max_abs_op(bf16_4_t{v.x, v.y}), max_abs_op(bf16_4_t{v.z, v.w}));
}

inline __device__ __hip_bfloat162 mul(__hip_bfloat162 v, __hip_bfloat16 scale) {
    return bf16hmul2(v, bf162bf162(scale));
}

inline __device__ bf16_4_t mul(bf16_4_t v, __hip_bfloat16 scale) {
    bf16_4_t n;
    n.x = mul(v.x, scale);
    n.y = mul(v.y, scale);
    return n;
}

inline __device__ bf16_8_t mul(bf16_8_t v, __hip_bfloat16 scale) {
    bf16_8_t n;
    n.x = mul(v.x, scale);
    n.y = mul(v.y, scale);
    n.z = mul(v.z, scale);
    n.w = mul(v.w, scale);
    return n;
}

inline __device__ void convert_to_fp8(fp8_4_t* v, const bf16_4_t u) {
    v[0] = fp8_4_t(u.x, u.y);
}

inline __device__ void convert_to_fp8(fp8_8_t* v, const bf16_8_t u) {
    v[0].x = fp8_2_t(u.x);
    v[0].y = fp8_2_t(u.y);
    v[0].z = fp8_2_t(u.z);
    v[0].w = fp8_2_t(u.w);
}

__forceinline__ __device__ float copysignf_pos(float a, float b)
{
    float r;
    r = __int_as_float(__float_as_int(a) | (__float_as_int(b) & 0x80000000));
    return r;
}

__inline__ __device__ float tanh_opt(float x)
{
#if (__CUDA_ARCH__ >= 750 && CUDART_VERSION >= 11000)
    float r;
    asm("tanh.approx.f32 %0,%1; \n\t" : "=f"(r) : "f"(x));
    return r;
#else
    const float exp_val = -1.f * fabs(2 * x);
    return copysignf_pos((1.0f - __expf(exp_val)) / (__expf(exp_val) + 1.0f), x);
#endif
}

static __device__ __forceinline__ __hip_bfloat162 silu(const __hip_bfloat162& val)
{
    return make_bfloat162((__hip_bfloat16)((float)val.x / (1.0f + __expf((float)-val.x))),
                          (__hip_bfloat16)((float)val.y / (1.0f + __expf((float)-val.y))));
}

inline __device__ bf16_8_t act_and_mul(bf16_8_t v, bf16_8_t u) {
    bf16_8_t n;
    n.x = bf16hmul2(silu(v.x), u.x);
    n.y = bf16hmul2(silu(v.y), u.y);
    n.z = bf16hmul2(silu(v.z), u.z);
    n.w = bf16hmul2(silu(v.w), u.w);
    return n;
}


template<typename T_S, bool COL_MAJOR_SCALE, int ELEM_PER_THREAD>
__global__ void computeFP8Quantize128Kernel(__hip_fp8_e4m3_fnuz*       fp8_output,
                                            T_S*                 quant_ptr,
                                            const __hip_bfloat16* weights,
                                            const int64_t        dim0,
                                            const int64_t        dim1,
                                            const int64_t        size) {
    const int64_t       global_idx     = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
    using InputElem = typename packed_type<__hip_bfloat16, ELEM_PER_THREAD>::type;
    using OutputElem = typename packed_type<__hip_fp8_e4m3_fnuz, ELEM_PER_THREAD>::type;
    auto weights_vec = reinterpret_cast<InputElem const*>(weights);
    auto output_vec = reinterpret_cast<OutputElem *>(fp8_output);

    if (global_idx * ELEM_PER_THREAD >= size) {
        return;
    }
    auto w8 = weights_vec[global_idx];
    float scale = cuda_max((float)1e-4, (float)max_abs_op(w8));
    static constexpr int THREADS_PER_ROW = 128 / ELEM_PER_THREAD;
#pragma unroll
    for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
        scale = max(scale, __shfl_xor_sync(0xFFFFFFFF, scale, mask, THREADS_PER_ROW));
    }
    scale = scale / FP8_E4M3_MAX;
    w8 = mul(w8, (__hip_bfloat16)(1 / scale));
    convert_to_fp8(output_vec + global_idx, w8);
    if (threadIdx.x % THREADS_PER_ROW == 0) {
        if constexpr (COL_MAJOR_SCALE) {
            const int64_t now_idx = global_idx / THREADS_PER_ROW;
            const int64_t row_idx = now_idx / dim1;
            const int64_t col_idx = now_idx % dim1;
            quant_ptr[col_idx * dim0 + row_idx] = scale;
        } else {
            quant_ptr[global_idx / THREADS_PER_ROW] = scale;
        }
    }
}

void invokeComputeFP8Quantize128(__hip_fp8_e4m3_fnuz*       fp8_output,
                                 float*               quant_ptr,
                                 const __hip_bfloat16* weights,
                                 const int64_t        dim0,
                                 const int64_t        dim1,
                                 const int64_t        size,
                                 bool                 col_major_scale,
                                 hipStream_t         stream) {
    RTP_LLM_CHECK(dim1 % 128 == 0);
    static constexpr int ELEM_PER_THREAD = 8;
    const int num_per_grid = CTA_SIZE / (128 / ELEM_PER_THREAD);
    dim3      grid((size / 128 + num_per_grid - 1) / num_per_grid);
    dim3      block(CTA_SIZE);
    if (col_major_scale) {
        computeFP8Quantize128Kernel<float, true, ELEM_PER_THREAD><<<grid, block, 0, stream>>>(fp8_output, quant_ptr, weights, dim0, dim1 / 128, size);
    } else {
        computeFP8Quantize128Kernel<float, false, ELEM_PER_THREAD><<<grid, block, 0, stream>>>(fp8_output, quant_ptr, weights, dim0, dim1, size);
    }
}

template<typename T_S, bool COL_MAJOR_SCALE, int ELEM_PER_THREAD>
__global__ void computeFP8ActivationAndQuantizeKernel(__hip_fp8_e4m3_fnuz*       fp8_output,
                                                      T_S*                 quant_ptr,
                                                      const __hip_bfloat16* gate_up_output,
                                                      const int64_t        dim0,
                                                      const int64_t        dim1) {
    const int64_t global_idx = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
    const int64_t size       = dim0 * dim1;
    const int64_t padded_dim0 = (dim0 + 63) / 64 * 64;
    if (global_idx * ELEM_PER_THREAD >= size) {
        return;
    }

    const int64_t row_idx = global_idx * ELEM_PER_THREAD / dim1;
    const int64_t col_idx = global_idx * ELEM_PER_THREAD % dim1;
    const int64_t row_stride = dim1 * 2;
    using InputElem = typename packed_type<__hip_bfloat16, ELEM_PER_THREAD>::type;
    using OutputElem = typename packed_type<__hip_fp8_e4m3_fnuz, ELEM_PER_THREAD>::type;
    auto weights_vec = reinterpret_cast<InputElem const*>(gate_up_output);
    auto output_vec = reinterpret_cast<OutputElem *>(fp8_output);

    auto gate8 = weights_vec[(row_idx * row_stride + col_idx) / ELEM_PER_THREAD], 
         up8   = weights_vec[(row_idx * row_stride + col_idx + dim1) / ELEM_PER_THREAD];

    auto w8 = act_and_mul(gate8, up8);
    float scale = cuda_max((float)1e-4, (float)max_abs_op(w8));
    static constexpr int THREADS_PER_ROW = 128 / ELEM_PER_THREAD;
#pragma unroll
    for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
        scale = max(scale, __shfl_xor_sync(0xFFFFFFFF, scale, mask, THREADS_PER_ROW));
    }
    scale = scale / FP8_E4M3_MAX;
    w8 = mul(w8, (__hip_bfloat16)(1 / scale));
    convert_to_fp8(output_vec + global_idx, w8);
    if (threadIdx.x % THREADS_PER_ROW == 0) {
        const int64_t dim = dim1 / 128;
        const int64_t now_idx = global_idx / THREADS_PER_ROW;
        const int64_t row_idx = now_idx / dim;
        const int64_t col_idx = now_idx % dim;
        quant_ptr[col_idx * padded_dim0 + row_idx] = scale;
    }
}

void computeFP8ActivationAndQuantize(__hip_fp8_e4m3_fnuz*       fp8_output,
                                     float*               quant_ptr,
                                     const __hip_bfloat16* weights,
                                     const int64_t        dim0,
                                     const int64_t        dim1,
                                     hipStream_t         stream) {
    RTP_LLM_CHECK(dim1 % 128 == 0);
    static constexpr int ELEM_PER_THREAD = 8;
    const int num_per_grid = CTA_SIZE / (128 / ELEM_PER_THREAD);
    const int size = dim0 * dim1;
    dim3      grid((size / 128 + num_per_grid - 1) / num_per_grid);
    dim3      block(CTA_SIZE);
    computeFP8ActivationAndQuantizeKernel<float, true, ELEM_PER_THREAD><<<grid, block, 0, stream>>>(fp8_output, quant_ptr, weights, dim0, dim1);
}

#endif // ENABLE_FP8
} // namespace common
} // namespace tensorrt_llm
